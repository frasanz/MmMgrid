#include "hip/hip_runtime.h"
#include <multigrid_kernel.cu>
#include <stdio.h>
#define N_MALLAS 12
#define BLOCK_SIZE 16
void g_imprime(Grid g,const char *);
void g_muestra(Grid g,const char *);
void multigrid(Grid *u, Grid *f, Grid *v, Grid *d, int m, float * max,float *def, float * hdef);

int main()
{
	int i;
	int dim;
	int size;
	int sizetotal=0;
	float max=1.0;
	float max_ant;

	/* Definimos los grids */
	Grid u[N_MALLAS];
	Grid f[N_MALLAS];
	Grid v[N_MALLAS];
	Grid d[N_MALLAS];

	/* un float para calcular el maximo del defecto de cada fila */
	float * def;  //En la GPU
	float * hdef; //En el host

	/* Reservamos memoria */
	for(i=0;i<N_MALLAS;i++)
	{
		dim  = (int)pow(2,i+1)+1;
		size = dim*dim;
		u[i].d = dim;
		f[i].d = dim;
		v[i].d = dim;
		d[i].d = dim;
		u[i].size = size;
		f[i].size = size;
		v[i].size = size;
		d[i].size = size;

		hipMalloc(&u[i].v,size*sizeof(float));
		hipMalloc(&f[i].v,size*sizeof(float));
		hipMalloc(&v[i].v,size*sizeof(float));
		hipMalloc(&d[i].v,size*sizeof(float));
		sizetotal+=4*size;
	}
	int m = N_MALLAS-1;
	dim = (int)pow(2,m+1)+1;
	size=dim*dim;
	hipMalloc(&def,size*sizeof(float));
	sizetotal+=size;
	hdef=(float*)malloc(size*sizeof(float));

	printf("Necesitamos %d Kb\n",sizetotal*sizeof(float)/1024);

	/* Para las llamadas a CUDA */
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid((dim+BLOCK_SIZE-1)/dimBlock.x,(dim+BLOCK_SIZE-1)/dimBlock.y);

	/* Inicializamos la malla de la función */
	cero<<<dimGrid, dimBlock>>>(f[m]);

	/* Inicializamos u[m] con valores "random" */
	cero<<<dimGrid, dimBlock>>>(u[m]);
	random<<<dimGrid, dimBlock>>>(u[m]);

	/* Principal */
	for(i=0;i<10;i++)
	{
		max_ant = max;
		max = 0.0;
		multigrid(&u[0],&f[0],&v[0],&d[0],m,&max,def,hdef);
		printf("[Iteracion #%d] nd=%e ratio=%f\n",i,max,max/max_ant);
	}


	/* Liberamos memoria */
	for(i=0;i<N_MALLAS;i++)
	{
		hipFree(u[i].v);
		hipFree(f[i].v);
		hipFree(v[i].v);
		hipFree(d[i].v);
	}
	hipFree(def);
	free(hdef);

	return 0;
}

void g_imprime(Grid g, const char *nombre)
{
	FILE * f;
	f=fopen(nombre,"w");
	int i,j;
	float *hg;
	hg=(float*)malloc(g.size*sizeof(float));
	hipMemcpy(hg,g.v,g.size*sizeof(float),hipMemcpyDeviceToHost);

	for(i=0;i<g.d;i++)
	{
		for(j=0;j<g.d;j++)
		{
			fprintf(f,"%d %d %f\n",i,j,hg[I(g.d,i,j)]);
		}
		fprintf(f,"\n");
	}
	fclose(f);
}
void g_muestra(Grid g, const char *nombre)
{
	int i,j;
	float *hg;
	hg=(float*)malloc(g.size*sizeof(float));
	hipMemcpy(hg,g.v,g.size*sizeof(float),hipMemcpyDeviceToHost);

	printf("%s=\n",nombre);
	for(i=0;i<g.d;i++)
	{
		for(j=0;j<g.d;j++)
		{
			printf("%f",hg[I(g.d,i,j)]);
		}
		printf("\n");
	}


}


void multigrid(Grid *u, Grid *f, Grid *v, Grid *d, int m, float * max,float *def, float * hdef)
{
	int dim;
	int dim_;
	int i;

	/* Primer caso, malla 0, solucion */
	if(m==0)
	{
		exacta<<<1,1>>>(u[m],f[m]);
	}
	else
	{
		/* Definiciones para las llamadas a CUDA */
		dim = (int)pow(2,m+1)+1;
		dim_= (int)pow(2,m)+1;
		dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
		dim3 dimGrid((dim+BLOCK_SIZE-1)/dimBlock.x,(dim+BLOCK_SIZE-1)/dimBlock.y);
		dim3 dimGrid_((dim_+BLOCK_SIZE-1)/dimBlock.x,(dim_+BLOCK_SIZE-1)/dimBlock.y);

		/* Ponemos a 0 las mallas necesarias */
		cero<<<dimGrid,dimBlock>>>(v[m]);
		cero<<<dimGrid,dimBlock>>>(d[m]);
		cero<<<dimGrid_,dimBlock>>>(u[m-1]);
		cero<<<dimGrid_,dimBlock>>>(f[m-1]);

		/* Suavizado R-N */
		suaviza_r<<<dimGrid,dimBlock>>>(u[m],f[m]);
		suaviza_n<<<dimGrid,dimBlock>>>(u[m],f[m]);

		/* Cálculo del defecto */
		defecto<<<dimGrid, dimBlock>>>(u[m],f[m],d[m]);

		/* Restringimos el defecto de d[m] a f[m-1] */
		restringe<<<dimGrid_,dimBlock>>>(d[m],f[m-1]);

		/* Rellamada a multigrid */
		multigrid(&u[0],&f[0],&v[0],&d[0],m-1,max,def,hdef);
		
		/* Interpolamos de u[m-1] a v[m] */
		interpola<<<dimGrid_,dimBlock>>>(u[m-1],v[m]);

		/* Sumamos */
		suma<<<dimGrid,dimBlock>>>(u[m],v[m]);

		/* Suavizado R-N */
		suaviza_r<<<dimGrid,dimBlock>>>(u[m],f[m]);
		suaviza_n<<<dimGrid,dimBlock>>>(u[m],f[m]);

		/* Si estamos en la malla superior, comprobamos el defecto */
		if(m==N_MALLAS-1)
		{
			defecto<<<dimGrid,dimBlock>>>(u[m],f[m],d[m]);
			dim3 dg((dim+BLOCK_SIZE-1)/dimBlock.x,1);
			dim3 db(BLOCK_SIZE,1);

			/* Calculamos el máximo de cada una de las filas */
			maxx<<<dg,db>>>(d[m],def);

			/* Copiamos el vector resultante al host */
			hipMemcpy(hdef,def,dim*dim*sizeof(float),hipMemcpyDeviceToHost);

			max[0]=0.0;
			for(i=0;i<dim;i++)
			{
				if(max[0]<hdef[i])
					max[0]=hdef[i];
			}
		}
	}
}

