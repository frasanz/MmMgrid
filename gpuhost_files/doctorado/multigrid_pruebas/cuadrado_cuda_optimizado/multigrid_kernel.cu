
#include <hip/hip_runtime.h>
#define I(d,i,j) (i)*(d)+(j) 
#define B(i) (i+1)
#define BLOCK_DIM 16

typedef struct{
	float *v;
	int d;
	int size;
} Grid;

__global__ void cero(Grid m)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<=m.d && j<=m.d)
		m.v[I(m.d,i,j)]=0.0;
}
__global__ void random(Grid m)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<m.d-1 && j<m.d-1 && i>0 && j>0) // Puntos interiores
		m.v[I(m.d,i,j)]=10.1+sinf(i+cosf(j));
}
__global__ void suaviza_r(Grid u,Grid f)
{
  __shared__ float bf[BLOCK_DIM][BLOCK_DIM];
  __shared__ float buo[BLOCK_DIM+2][BLOCK_DIM+2];

  float h2 = pow(1.0/(u.d-1),2);
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  int n=threadIdx.x;
  int m=threadIdx.y;
  if(i<=u.d-1 && j<=u.d-1 && i>=0 && j>=0) // Carga chapucera en shared
  {
    bf[threadIdx.x][threadIdx.y] = f.v[I(u.d,i  ,j  )];
    buo[threadIdx.x+1][threadIdx.y+1] = u.v[I(u.d,i,j  )];
    if(threadIdx.x==0 && i-1>=0) //Si el limite sup del bloque está, lo cargamos
      buo[threadIdx.x][threadIdx.y+1] = u.v[I(u.d,i-1,j  )];
    if(threadIdx.x==(u.d-2)%BLOCK_DIM && i+1<=u.d-1)
      buo[threadIdx.x+2][threadIdx.y+1] = u.v[I(u.d,i+1,j  )];
    if(threadIdx.y==0 && j-1>=0) //Si el limite izq del bloque está, lo cargamos
      buo[threadIdx.x+1][threadIdx.y] = u.v[I(u.d,i,j-1  )];
    if(threadIdx.y==(u.d-2)%BLOCK_DIM && j+1<=u.d-1)
      buo[threadIdx.x+1][threadIdx.y+2] = u.v[I(u.d,i,j+1  )];
  }
  __syncthreads();

  if(i<u.d-1 && j<u.d-1 && i>0 && j>0) // Puntos interiores
  {
    if((i+j)%2==0)
      u.v[I(u.d,i,j)]=0.25*(bf[n][m]*h2
                           +buo[B(n-1)][B(m)]
                           +buo[B(n+1)][B(m)]
                           +buo[B(n)][B(m-1)]
                           +buo[B(n)][B(m+1)]);
  }
}
__global__ void suaviza_n(Grid u,Grid f)
{
  __shared__ float bf[BLOCK_DIM][BLOCK_DIM];
  __shared__ float buo[BLOCK_DIM+2][BLOCK_DIM+2];

  float h2 = pow(1.0/(u.d-1),2);
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  int n=threadIdx.x;
  int m=threadIdx.y;
  if(i<=u.d-1 && j<=u.d-1 && i>=0 && j>=0) // Carga chapucera en shared
  {
    bf[threadIdx.x][threadIdx.y] = f.v[I(u.d,i  ,j  )];
    buo[threadIdx.x+1][threadIdx.y+1] = u.v[I(u.d,i,j  )];
    if(threadIdx.x==0 && i-1>=0) //Si el limite sup del bloque está, lo cargamos
      buo[threadIdx.x][threadIdx.y+1] = u.v[I(u.d,i-1,j  )];
    if(threadIdx.x==(u.d-2)%BLOCK_DIM && i+1<=u.d-1)
      buo[threadIdx.x+2][threadIdx.y+1] = u.v[I(u.d,i+1,j  )];
    if(threadIdx.y==0 && j-1>=0) //Si el limite izq del bloque está, lo cargamos
      buo[threadIdx.x+1][threadIdx.y] = u.v[I(u.d,i,j-1  )];
    if(threadIdx.y==(u.d-2)%BLOCK_DIM && j+1<=u.d-1)
      buo[threadIdx.x+1][threadIdx.y+2] = u.v[I(u.d,i,j+1  )];
  }
  __syncthreads();

  if(i<u.d-1 && j<u.d-1 && i>0 && j>0) // Puntos interiores
  {
    if((i+j)%2==1)
      u.v[I(u.d,i,j)]=0.25*(bf[n][m]*h2
                           +buo[B(n-1)][B(m)]
                           +buo[B(n+1)][B(m)]
                           +buo[B(n)][B(m-1)]
                           +buo[B(n)][B(m+1)]);
  }

}
__global__ void defecto(Grid u, Grid f, Grid d)
{
  float h2 = pow(1.0/(u.d-1),2);

  __shared__ float bu[BLOCK_DIM+2][BLOCK_DIM+2];
  __shared__ float bf[BLOCK_DIM][BLOCK_DIM];

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int n=threadIdx.x;
  int m=threadIdx.y;
  if(i<=u.d-1 && j<=u.d-1 && i>=0 && j>=0) // Carga chapucera en shared
  {
    bf[threadIdx.x][threadIdx.y] = f.v[I(u.d,i  ,j  )];
    bu[threadIdx.x+1][threadIdx.y+1] = u.v[I(u.d,i,j  )];
    if(threadIdx.x==0 && i-1>=0) //Si el limite sup del bloque está, lo cargamos
      bu[threadIdx.x][threadIdx.y+1] = u.v[I(u.d,i-1,j  )];
    if(threadIdx.x==(u.d-2)%BLOCK_DIM && i+1<=u.d-1)
      bu[threadIdx.x+2][threadIdx.y+1] = u.v[I(u.d,i+1,j  )];
    if(threadIdx.y==0 && j-1>=0) //Si el limite izq del bloque está, lo cargamos
      bu[threadIdx.x+1][threadIdx.y] = u.v[I(u.d,i,j-1  )];
    if(threadIdx.y==(u.d-2)%BLOCK_DIM && j+1<=u.d-1)
      bu[threadIdx.x+1][threadIdx.y+2] = u.v[I(u.d,i,j+1  )];
  }
  __syncthreads();

  if(i<u.d-1 && j<u.d-1 && i>0 && j>0) // Puntos interiores
  {
    d.v[I(u.d,i,j)]= bf[threadIdx.x][threadIdx.y]
                  -(4*bu[B(n)][B(m)]
                     -bu[B(n-1)][B(m)]
                     -bu[B(n+1)][B(m)]
                     -bu[B(n)][B(m-1)]
                     -bu[B(n)][B(m+1)])/h2;
  }
}
__global__ void restringe(Grid sup, Grid in)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if(i<in.d-1 && j<in.d-1 && i>0 && j>0) // Puntos interiores
  {
    in.v[I(in.d,i,j)] = (4* sup.v[I(sup.d,2*i  ,2*j  )]
                        +2*(sup.v[I(sup.d,2*i-1,2*j  )]
                           +sup.v[I(sup.d,2*i+1,2*j  )]
                           +sup.v[I(sup.d,2*i  ,2*j-1)]
                           +sup.v[I(sup.d,2*i  ,2*j+1)])
                           +sup.v[I(sup.d,2*i-1,2*j-1)]
                           +sup.v[I(sup.d,2*i-1,2*j+1)]
                           +sup.v[I(sup.d,2*i+1,2*j-1)]
                           +sup.v[I(sup.d,2*i+1,2*j+1)])/16;
  }
}

__global__ void exacta(Grid u, Grid f)
{
	u.v[I(u.d,1,1)]=f.v[I(u.d,1,1)]/16;
}
__global__ void interpola(Grid u, Grid v)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<u.d && j<u.d)
	{
		v.v[I(v.d,2*i,2*j)] = u.v[I(u.d,i,j)];
		if(2*i+1<v.d)
			v.v[I(v.d,2*i+1,2*j)]=(u.v[I(u.d,i,j)]+u.v[I(u.d,i+1,j)])/2;
		if(2*j+1<v.d)
			v.v[I(v.d,2*i,2*j+1)]=(u.v[I(u.d,i,j)]+u.v[I(u.d,i,j+1)])/2;
		if(2*i+1<v.d && 2*j+1<v.d)
			v.v[I(v.d,2*i+1,2*j+1)]=(u.v[I(u.d,i,j)]+u.v[I(u.d,i+1,j)]+u.v[I(u.d,i,j+1)]+u.v[I(u.d,i+1,j+1)])/4;
	}
}
__global__ void suma(Grid u, Grid v)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  __shared__ float bv[BLOCK_DIM][BLOCK_DIM];
  if(i<u.d && j<u.d)
  {
     bv[threadIdx.x][threadIdx.y]=v.v[I(u.d,i,j)];
  }
  __syncthreads();
  if(i<u.d && j<u.d)
  {
    u.v[I(u.d,i,j)]+=bv[threadIdx.x][threadIdx.y];
  }
}

__global__ void maxx(Grid d,float * def)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	def[i]=0.0;
	for(j=0;j<d.d;j++)
	{
		if(abs(d.v[I(d.d,i,j)])>def[i])
		{
			def[i]=abs(d.v[I(d.d,i,j)]);
		}
	}
}


