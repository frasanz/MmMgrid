#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <wilson_kernel.cu>

#define BLOCK_SIZE 32 
Representacion mul(Representacion a, Representacion b, u64 p)
{
	Representacion mul;
	mul.r2=(a.r2*b.r2)%p;
	mul.r1=(a.r2*b.r2)/p;
	mul.r1=(mul.r1+a.r1*b.r2+b.r1*a.r2)%p;
	return mul;
}

int main(int argc, char *argv)
{
	u64 primo=50000003;
	Representacion * salida;
	Representacion * g_salida;
	int i;

	/* Reservamos memoria */
	salida = (Representacion *)malloc(BLOCK_SIZE * sizeof(Representacion));
	hipMalloc((void**)&g_salida,BLOCK_SIZE*sizeof(Representacion));

	/* A cero */
	//cero<<<1,BLOCK_SIZE>>>(g_salida);
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);

	/* Calculamos */
	fac_mod_parll<<<1,dimBlock>>>(g_salida,primo,primo/(BLOCK_SIZE*BLOCK_SIZE-1));

	/* Copiamos la solucion */
	hipMemcpy(salida,g_salida,BLOCK_SIZE*sizeof(Representacion),hipMemcpyDeviceToHost);

	/* Los 256 los multiplicamos localmente */
	Representacion sol;
	sol.r1=1;
	sol.r2=1;
	for(i=0;i<BLOCK_SIZE;i++)
		sol=mul(sol,salida[i],primo);

	/* Imprimimos por pantalla */
	printf("%lld %lld\n",sol.r1,sol.r2);

	/* Liberamos memoria */
	hipFree(g_salida);
	free(salida);

}
