
#include <hip/hip_runtime.h>
typedef unsigned long long u64;
typedef struct {
	u64 r1;
	u64 r2;
} Representacion;

__device__ Representacion g_get_representacion(u64 a, u64 p)
{
	Representacion re;
	re.r1=a/p;
	re.r2=a%p;
	return re;
}
__device__ Representacion g_mul(Representacion a, Representacion b, u64 p)
{
	Representacion mul;
	mul.r2=(a.r2*b.r2)%p;
	mul.r1=(a.r2*b.r2)/p;
	mul.r1=(mul.r1+a.r1*b.r2+b.r1*a.r2)%p;
	return mul;
}
__global__ void cero(Representacion *num)
{
	num[threadIdx.x].r1=1;
	num[threadIdx.x].r2=1;
}

__global__ void fac_mod_parll(Representacion * num, u64 primo, u64 espacio)
{
	u64 i;
	u64 max;
	__shared__ Representacion mult;
	mult.r1=1;
	mult.r2=1;
	if((threadIdx.x+1)*espacio+1 > primo-1)
		max=primo-1;
	else
		max=(threadIdx.x+1)*espacio+1;
	for(i=threadIdx.x*espacio+2;i<=max;i++)
	{
		mult=g_mul(mult,g_get_representacion(i,primo),primo);
	}
	num[threadIdx.x]=mult;
}
