#include "hip/hip_runtime.h"
#include <multigrid_kernel.cu>
#include <stdio.h>
#include <stdlib.h>
#include "multigrid.h"
#include "auxiliares.h"
#include "io.h"
#include "operador.h"
#define BLOCK_SIZE 16

int main(int argc, char **argv)
{

	int i;
	int n_mallas=5;
	const char *uso=
		"\nUso: %s \n\t -m número de mallas \
		 \n\t -tr archivo de triangulos \n\n";
	if((argc>1) && (!strcmp(argv[1], "-h") || !strcmp(argv[1], "--help"))) {
		printf (uso, argv[0]);
		exit(1);
	}
	char *archivo_triangulo="triangulo";

	for(i=1; i<argc;i++)
	{
		if(!strcmp(argv[i],"-m"))
			n_mallas=atoi(argv[++i]);
		else if(!strcmp(argv[i],"-tr"))
			archivo_triangulo=argv[++i];
	}

	int dim;
	int size;
	int size_total=0;
	int malla_actual=n_mallas;	

	/* El triangulo a usar */
	Triangulo tr;

	/* Definimos las mallas */
	Grid u[n_mallas];
	Grid f[n_mallas];
	Grid v[n_mallas];
	Grid d[n_mallas];

	/* Para el cálculo del máximo del defecto */
	double * h_def; /* En el Host */ 
	double max;
	double max_anterior;

	/* Reservamos la memoria en la GPU*/
	for(i=2;i<=n_mallas;i++)
	{
		dim  = pow(2,i)+1;	/* Dim es el número de elementos de la "diagonal" */
		size = dim*dim;			/* Reservamos una matriz entera, aunque sólo usaremos por debajo de la diagonal */
		u[i].dim = dim;
		f[i].dim = dim;
		v[i].dim = dim;
		d[i].dim = dim;
		u[i].size = size;
		f[i].size = size;
		v[i].size = size;
		d[i].size = size;

		hipMalloc(&u[i].v,size*sizeof(double));
		hipMalloc(&f[i].v,size*sizeof(double));
		hipMalloc(&v[i].v,size*sizeof(double));
		hipMalloc(&d[i].v,size*sizeof(double));

		hipMalloc(&u[i].op,9*sizeof(double));
		hipMalloc(&f[i].op,9*sizeof(double));
		hipMalloc(&v[i].op,9*sizeof(double));
		hipMalloc(&d[i].op,9*sizeof(double));

		size_total=size_total+4*size+4*9;
	}
	h_def=(double*)malloc(dim*dim*sizeof(double));

	printf("Hemos reservado %d kb en la GPU\n",size_total*sizeof(double)/1024);

	/* Creacion de el operador */
	double  op[3][3]={{0,0,0},{0,0,0,},{0,0,0}};
	double  op_es[3][3]={{0,0,0},{0,0,0},{0,0,0}};
	tr=lee_fichero_triangulo(archivo_triangulo);
	crea_operador(tr,&op[0][0]);
	
	/* Y escalamos el operador para cada malla */
	for(i=2;i<=n_mallas;i++)
	{
		iguala_matriz(&op[0][0],&op_es[0][0],3);
		escala_matriz(&op_es[0][0],3,pow((u[i].dim-1),2));
		hipMemcpy(u[i].op,&op_es[0][0],9*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(f[i].op,&op_es[0][0],9*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(v[i].op,&op_es[0][0],9*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy(d[i].op,&op_es[0][0],9*sizeof(double),hipMemcpyHostToDevice);
	}
//	imprime_grid(u[malla_actual],"grid");
	

	/* Datos para la llamada a CUDA */
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid((u[malla_actual].dim+BLOCK_SIZE-1)/dimBlock.x,(u[malla_actual].dim+BLOCK_SIZE-1)/dimBlock.y);

	/* Inicializamos la malla de la función f */
	cero<<<dimGrid,dimBlock>>>(f[malla_actual]);
	random<<<dimGrid,dimBlock>>>(u[malla_actual]);

	/* Principal */
	for(i=0;i<10;i++)
	{
		max_anterior = max;
		max=0.0;
		multigrid(&u[0],&f[0],&v[0],&d[0],malla_actual,&max,h_def,n_mallas);
		printf("[Iteracion #%d]\n",i);
		printf("max=%0.10e ratio=%f\n",max,max/max_anterior);
	}


	/* Liberacion de memoria */
	for(i=2;i<n_mallas;i++)
	{
		hipFree(&u[i].v);
		hipFree(&v[i].v);
		hipFree(&f[i].v);
		hipFree(&d[i].v);
		hipFree(&u[i].op);
		hipFree(&v[i].op);
		hipFree(&f[i].op);
		hipFree(&d[i].op);

	}
	free(h_def);
	return 0;
}

void multigrid(Grid *u, Grid *f, Grid *v, Grid *d, int m, double * max,double * h_def,int n_mallas)
{
	int dim;
	int dim_;
	int i,j;
	if(m==2) /* Resolver exacta */
	{
	}
	else
	{
		/* Datos para la llamada a CUDA */
		dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
		dim3 dimGrid((u[m].dim+BLOCK_SIZE-1)/dimBlock.x,(u[m].dim+BLOCK_SIZE-1)/dimBlock.y);
		dim3 dimGrid_((u[m-1].dim+BLOCK_SIZE-1)/dimBlock.x,(u[m-1].dim+BLOCK_SIZE-1)/dimBlock.y);

		/* Ponemos a 0 las mallas necesarias */
		cero<<<dimGrid,dimBlock>>>(v[m]);
		cero<<<dimGrid,dimBlock>>>(d[m]);
		cero<<<dimGrid_,dimBlock>>>(u[m-1]);
		cero<<<dimGrid_,dimBlock>>>(f[m-1]);

		/* Suavizamos */
		suaviza<<<dimGrid,dimBlock>>>(u[m],f[m],0);
		suaviza<<<dimGrid,dimBlock>>>(u[m],f[m],1);
		suaviza<<<dimGrid,dimBlock>>>(u[m],f[m],2);

		/* Calculamos el defecto */
		defecto<<<dimGrid,dimBlock>>>(u[m],f[m],d[m]);

		/* Lo restringimos */
		restringe<<<dimGrid_,dimBlock>>>(d[m],f[m-1]);

		/* Rellamamos a multigrid */
		multigrid(&u[0],&f[0],&v[0],&d[0],m-1,max,h_def,n_mallas);

		/* Interpolamos */
		interpola<<<dimGrid_,dimBlock>>>(u[m-1],v[m]);

		/* Sumamos */
		suma<<<dimGrid,dimBlock>>>(u[m],v[m]);

		/* Suavizamos */
		suaviza<<<dimGrid,dimBlock>>>(u[m],f[m],0);
		suaviza<<<dimGrid,dimBlock>>>(u[m],f[m],1);
		suaviza<<<dimGrid,dimBlock>>>(u[m],f[m],2);

		/* Si estamos en la malla superior, comprobamos el defecto */
		if(m==n_mallas)
		{
			printf("En comprobación...\n");
			/* Calculamos el defecto */
			defecto<<<dimGrid,dimBlock>>>(u[m],f[m],d[m]);

			/* Copiamos al host el valor para calcular su máximo */
			hipMemcpy(h_def,d[m].v,d[m].dim*d[m].dim*sizeof(double),hipMemcpyDeviceToHost);

			max[0] = 0.0;
			for(i=1;i<d[m].dim;i++)
			{
				for(j=1;j<i;j++)
				{
					if(max[0]<h_def[i])
						max[0]=h_def[i];
				}
			}
		}
	}
}
