/*
 * =====================================================================================
 *
 *       Filename:  auxiliares.c
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  30/11/10 19:42:36
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *        Company:  
 *
 * =====================================================================================
 */
#include "auxiliares.h"
#include "multigrid.h"
#include <stdio.h>

void imprime_grid(Grid g, const char *nombre)
{
	int i,j;
	double *hg;
	double *opg;
	/* Reservamos memoria para traer el grid de la GPU al host */
	hg = (double *)malloc(g.size*sizeof(double));
	opg= (double *)malloc(9*sizeof(double));
	hipMemcpy(hg,g.v,g.size*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(opg,g.op,9*sizeof(double),hipMemcpyDeviceToHost);
	printf("%s(tamaño=%d)=\n",nombre,g.size);
	for(i=0;i<g.dim;i++)
	{
		for(j=0;j<g.dim;j++)
		{
			printf("(%d %d)%e ",i,j,hg[I(g.dim,i,j)]);
		}
		printf("\n");
	}
	printf("operador=\n");
	for(i=0;i<9;i++)
		printf("%e ",opg[i]);
	printf("\n");
	free(opg);
	free(hg);
}

