#include "hip/hip_runtime.h"
#include "multigrid.h"

__global__ void cero(Grid m)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<m.dim && j<=i)
		m.v[I(m.dim,i,j)]=0.0;
}
__global__ void random(Grid m)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i<m.dim && j<m.dim)
		m.v[I(m.dim,i,j)]=0.0;
	if(i<m.dim-1 && j<i && i>0 && j>0)
		m.v[I(m.dim,i,j)]=10.2*sinf(i+cosf(j));
}
__global__ void suaviza(Grid u, Grid f,int color)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i>0 && j> 0  && i < u.dim-1 && j<i)
	{
		if((i+j)%3==color)
		{

			u.v[I(u.dim,i,j)]=(f.v[I(u.dim,i,j)]-(
												 u.op[0]*u.v[I(u.dim,i-1,j-1)]
												+u.op[1]*u.v[I(u.dim,i-1,j)]
												+u.op[3]*u.v[I(u.dim,i,j-1)]
												+u.op[5]*u.v[I(u.dim,i,j+1)]
												+u.op[6]*u.v[I(u.dim,i+1,j)]
												+u.op[7]*u.v[I(u.dim,i+1,j+1)]))/u.op[4];
		}
	}
}
__global__ void defecto(Grid u, Grid f, Grid d)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i>0 && j> 0  && i < u.dim-1 && j<i)
	{
		d.v[I(d.dim,i,j)]=(f.v[I(d.dim,i,j)]-(
			 u.op[0]*u.v[I(u.dim,i-1,j-1)]
			+u.op[1]*u.v[I(u.dim,i-1,j  )]
			+u.op[3]*u.v[I(u.dim,i  ,j-1)]
			+u.op[4]*u.v[I(u.dim,i  ,j  )]
			+u.op[5]*u.v[I(u.dim,i  ,j+1)]
			+u.op[7]*u.v[I(u.dim,i+1,j  )]
			+u.op[8]*u.v[I(u.dim,i+1,j+1)]
		));
	}
}
__global__ void restringe(Grid sup, Grid inf)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i>0 && j> 0  && i < inf.dim-1 && j<i)
	{
		inf.v[I(inf.dim,i,j)]=(sup.v[I(sup.dim,2*i  ,2*j  )]+
				0.5*(sup.v[I(sup.dim,2*i-1,2*j-1)]+
					sup.v[I(sup.dim,2*i-1,2*j  )]+
					sup.v[I(sup.dim,2*i  ,2*j-1)]+
					sup.v[I(sup.dim,2*i  ,2*j+1)]+
					sup.v[I(sup.dim,2*i+1,2*j  )]+
					sup.v[I(sup.dim,2*i+1,2*j+1)]))/4;
	}
}
__global__ void interpola(Grid inf, Grid sup)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i>0 && j> 0  && i < inf.dim-1 && j<i)
	{
		sup.v[I(sup.dim,2*i  ,2*j  )] = inf.v[I(inf.dim,i  ,j  )];
		sup.v[I(sup.dim,2*i-1,2*j-1)] =(inf.v[I(inf.dim,i  ,j  )] + inf.v[I(inf.dim,i-1,j-1)] )/2;
		sup.v[I(sup.dim,2*i-1,2*j  )] =(inf.v[I(inf.dim,i  ,j  )] + inf.v[I(inf.dim,i-1,j  )] )/2;
		sup.v[I(sup.dim,2*i  ,2*j-1)] =(inf.v[I(inf.dim,i  ,j  )] + inf.v[I(inf.dim,i  ,j-1)] )/2;
		sup.v[I(sup.dim,2*i  ,2*j+1)] =(inf.v[I(inf.dim,i  ,j  )] + inf.v[I(inf.dim,i  ,j+1)] )/2;
		sup.v[I(sup.dim,2*i+1,2*j  )] =(inf.v[I(inf.dim,i  ,j  )] + inf.v[I(inf.dim,i+1,j  )] )/2;
		sup.v[I(sup.dim,2*i+1,2*j  )] =(inf.v[I(inf.dim,i  ,j  )] + inf.v[I(inf.dim,i+1,j  )] )/2;
	}
}
__global__ void suma(Grid u, Grid v)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i>0 && j> 0 && i< u.dim-1 && j< u.dim-1)
	{
		u.v[I(u.dim,i,j)]=u.v[I(u.dim,i,j)]+v.v[I(v.dim,i,j)];
	}
}
