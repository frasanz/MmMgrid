#include "hip/hip_runtime.h"
#include <multigrid_kernel.cu>
#include <stdio.h>
#define N_MALLAS 4
#define BLOCK_SIZE 16

void g_imprime(Grid g);
void multigrid(Grid *u, Grid *f, Grid *v, Grid *d,int nivel, float * op, float *max);

int main()
{
	int i;
	int dim;
	int size;
	float max;
	float max_ant;

	/* Definicion de las Mallas */
	Grid u[N_MALLAS];
	Grid f[N_MALLAS];
	Grid v[N_MALLAS];
	Grid d[N_MALLAS];

	/* Reservamos la memoria */
	for(i=2;i<N_MALLAS;i++)
	{
		dim=pow(2,i)+1; //Dim es el número de elementos de la "diagonal"
		size=((dim-1)*(dim-1)+3*(dim-1))/2+1;
		u[i].dim=dim;
		f[i].dim=dim;
		v[i].dim=dim;
		d[i].dim=dim;
		u[i].size=size;
		f[i].size=size;
		v[i].size=size;
		d[i].size=size;

		hipMalloc(&u[i].v,size*sizeof(float));
		hipMalloc(&f[i].v,size*sizeof(float));
		hipMalloc(&v[i].v,size*sizeof(float));
		hipMalloc(&d[i].v,size*sizeof(float));
	}
	/* Para el operador en el device */
	float * d_op;
	hipMalloc(&d_op,9*sizeof(float));


	/* Para la llamada a CUDA */
	int m=N_MALLAS-1;
	dim=(int)pow(2,m)+1;
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid((dim+BLOCK_SIZE-1)/dimBlock.x,(dim+BLOCK_SIZE-1)/dimBlock.y);
	printf("%d %d %d %d\n",dimBlock.x, dimBlock.y,dimGrid.x,dimGrid.y);

	/* Inicializamos la malla de la funcion */
	inicializa_f<<<dimGrid,dimBlock>>>(f[m]);

	/* Initialize u with random values */
	random<<<dimGrid,dimBlock>>>(u[m]);

	/* Principal */
	for(i=0;i<1;i++)
	{
		max_ant=max;
		max=0.0;
		multigrid(&u[0],&f[0],&v[0],&d[0],m,d_op,&max);
		printf("Iteracion %d nd=%f ratio=%f\n",i,max,max/max_ant);
	}

	/* Liberamos la memoria */
	for(i=2;i<N_MALLAS;i++)
	{
		hipFree(&u[i].v);
		hipFree(&f[i].v);
		hipFree(&v[i].v);
		hipFree(&d[i].v);

	}
	hipFree(d_op);

	return 0;
}

/* This function prints a grid allocated in GPU */
void g_imprime(Grid g)
{
	float * dg;
	int i,j;
	size_t size=((g.dim-1)*(g.dim-1)+3*(g.dim-1))/2+1;
	dg=(float*)malloc(size*sizeof(float));
	hipMemcpy(dg,g.v,size*sizeof(float),hipMemcpyDeviceToHost);

	/* Mostramos por pantalla */
	printf("Grid size=%d dim=%d\n",size,g.dim);
	for(i=0;i<g.dim;i++)
	{
		for(j=0;j<=i;j++)
			printf("%f ",dg[IDT(i,j)]);
		printf("\n");
	}
	printf("\n");

	free(dg);
}
void multigrid(Grid *u, Grid *f, Grid *v, Grid *d,int m,float * d_op, float *max)
{
	int dim;
	int dim_;
	int i;
	/* Definimos h^2 */
	float h2=pow(u[m].dim-1,2);

	/* Definimos un operador (copiado de otro sitio) */
	float operador[9]={0.0,-1.0*h2,0.0,-1.0*h2,4.0*h2,-1.0*h2,0.0,-1.0*h2,0.0};
	hipMemcpy(d_op,&operador[0],9*sizeof(float),hipMemcpyHostToDevice);
	/* Para el operador en el device */

	if(m==2)
	{
		/* Dimension, necesitamos bajar la f al host para resolver el sistema lineal */
		float * hf;
		float * hu;
		dim=(int)pow(2,m)+1;
		size_t size=((f[m].dim-1)*(f[m].dim-1)+3*(f[m].dim-1))/2+1;
		hf=(float*)malloc(size*sizeof(float));
		hipMemcpy(hf,f[m].v,size*sizeof(float),hipMemcpyDeviceToHost);
		hu=(float*)malloc(size*sizeof(float));
		for(i=0;i<size;i++)
			hu[i]=0.0;

		/* Construimos el sistema a resolver */
		double A[3][3];
		A[0][0]=operador[4];
		A[0][1]=operador[7];
		A[0][2]=operador[8];
		A[1][0]=operador[2];
		A[1][1]=operador[4];
		A[1][2]=operador[5];
		A[2][0]=operador[0];
		A[2][1]=operador[3];
		A[2][2]=operador[4];

		double B[3];
		B[0]=hf[IDT(2,1)];
		B[1]=hf[IDT(3,1)];
		B[2]=hf[IDT(3,2)];

		/* Hacemos eliminación gausiana */
		A[1][1]=A[1][1]-A[0][1]*A[1][0]/A[0][0];
		A[1][2]=A[1][2]-A[0][2]*A[1][0]/A[0][0];
		B[1]=B[1]-B[0]*A[1][0]/A[0][0];
		A[2][1]=A[2][1]-A[0][1]*A[2][0]/A[0][0];
		A[2][2]=A[2][2]-A[0][2]*A[2][0]/A[0][0];
		B[2]=B[2]-B[0]*A[2][0]/A[0][0];
		A[2][2]=A[2][2]-A[1][2]*A[2][1]/A[1][1];
		B[2]=B[2]-B[1]*A[2][1]/A[1][1];

		/* Resolvemos */
		hu[IDT(3,2)]=B[2]/A[2][2];
		hu[IDT(3,1)]=(B[1]-A[1][2]*hu[IDT(3,2)])/A[1][1];
		hu[IDT(2,1)]=(B[0]-A[0][2]*hu[IDT(3,2)]-A[0][1]*hu[IDT(3,1)])/A[0][0];

		/* Subimos la solución a la GPU */
		hipMemcpy(u[m].v,hu,size*sizeof(float),hipMemcpyHostToDevice);
		free(hf);
	}
	else
	{
		/* Para la llamada a CUDA */
		dim=(int)pow(2,m)+1;
		dim_=(int)pow(2,m-1)+1;
		dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
		dim3 dimGrid((dim+BLOCK_SIZE-1)/dimBlock.x,(dim+BLOCK_SIZE-1)/dimBlock.y);
		dim3 dimGrid_((dim_+BLOCK_SIZE-1)/dimBlock.x,(dim_+BLOCK_SIZE-1)/dimBlock.y);

		/* Ponemos a 0 las mallas necesarias */
		cero<<<dimGrid,dimBlock>>>(v[m]);
		cero<<<dimGrid,dimBlock>>>(d[m]);
		cero<<<dimGrid,dimBlock>>>(u[m-1]);
		cero<<<dimGrid,dimBlock>>>(f[m-1]);

		/* Suavizamos tres colores */
		g_imprime(u[m]);
		suaviza_r<<<dimGrid,dimBlock>>>(u[m],f[m],d_op);
		suaviza_g<<<dimGrid,dimBlock>>>(u[m],f[m],d_op);
		suaviza_b<<<dimGrid,dimBlock>>>(u[m],f[m],d_op);
		g_imprime(u[m]);

		/* Calculamos el defecto */
		defecto<<<dimGrid,dimBlock>>>(u[m],f[m],d[m],d_op);
		g_imprime(d[m]);

		/* Restringimos el defecto (de d[m] a f[m-1]) */
		restringe<<<dimGrid_,dimBlock>>>(d[m],f[m-1]);
		g_imprime(f[m-1]);

		/* Rellamamos a multigrid */
		multigrid(&u[0],&f[0],&v[0],&d[0],m-1,d_op,max);
		g_imprime(u[m-1]);

		/* Interpolate from u[m-1] to v[m] */
		interpola<<<dimGrid_,dimBlock>>>(u[m-1],v[m]);
		g_imprime(v[m]);

		/* Sumamos */
		suma<<<dimGrid,dimBlock>>>(u[m],v[m]);
		g_imprime(u[m]);

		/* Post-suavizamos tres colores */
		suaviza_r<<<dimGrid,dimBlock>>>(u[m],f[m],d_op);
		suaviza_g<<<dimGrid,dimBlock>>>(u[m],f[m],d_op);
		suaviza_b<<<dimGrid,dimBlock>>>(u[m],f[m],d_op);
		g_imprime(u[m]);

		/* Si estamos en la malla superior,comprobamos cómo va el defecto */
		if(m==N_MALLAS-1) 
		{
			defecto<<<dimGrid,dimBlock>>>(u[m],f[m],d[m],d_op);
			g_imprime(d[m]);
			float * def;
			size_t size=((f[m].dim-1)*(f[m].dim-1)+3*(f[m].dim-1))/2+1;
			def=(float*)malloc(size*sizeof(float));
			hipMemcpy(def,d[m].v,size*sizeof(float),hipMemcpyDeviceToHost);
			for(i=0;i<size;i++)
			{
				if(max[0]<fabs(def[i]))
					max[0]=fabs(def[i]);
			}
			free(def);
		}



	}
}
