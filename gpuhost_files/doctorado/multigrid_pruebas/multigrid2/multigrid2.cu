#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <multigrid2_kernel.cu>

#define BLOCK_SIZE 16
#define N_MALLAS 12 
void imprime_g(Malla);
void multigrid(Malla *, Malla *, Malla *, Malla *, int, double *, double *,double *);

int main(int argc, char **argv)
{

	int i;
	int dim;
	int nivel;
	Malla * mu;
	Malla * mv;
	Malla * mf;
	Malla * md;
	mu = (Malla *)malloc(N_MALLAS*sizeof(Malla));
	mv = (Malla *)malloc(N_MALLAS*sizeof(Malla));
	mf = (Malla *)malloc(N_MALLAS*sizeof(Malla));
	md = (Malla *)malloc(N_MALLAS*sizeof(Malla));

	/* Para calcular los defectos */
	double * def_fila_g;
	double * def_fila;
	double nd;
	double nda;
	dim  =(int)pow(2.0,N_MALLAS)+1;
	hipMalloc((void**)&def_fila_g,dim*sizeof(double));
	def_fila=(double *)malloc(dim*sizeof(double));


	printf("Reservando memoria...\n");
	for(i=0;i<N_MALLAS;i++)
	{
		dim = (int)pow(2.0,i+1)+1;
		mu[i].dim=dim;
		mv[i].dim=dim;
		mf[i].dim=dim;
		md[i].dim=dim;
		hipMalloc((void**)&mu[i].v,dim*dim*sizeof(double));
		hipMalloc((void**)&mv[i].v,dim*dim*sizeof(double));
		hipMalloc((void**)&mf[i].v,dim*dim*sizeof(double));
		hipMalloc((void**)&md[i].v,dim*dim*sizeof(double));
	}

	nivel=N_MALLAS-1;
	dim  =(int)pow(2.0,N_MALLAS)+1;
	/* Parámtros de CUDA */
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(((int)dim+BLOCK_SIZE-1)/dimBlock.x,((int)dim+BLOCK_SIZE-1)/dimBlock.y);
	printf("%d %d\n",dimGrid.x,dimGrid.y);

	/* Definimos la malla de la funcion en la GPU */
	inicializa_f<<<dimGrid,dimBlock>>>(mf[nivel]);

	/* Inicializamos a 0 la malla de la solución */
	cero<<<dimGrid,dimBlock>>>(mu[nivel]);

	/* Llamamos a multigrid */
	printf("Llamada a multigrid...\n");
	for(i=0;i<10;i++)
	{
		nda=nd;
		nd=0.0;
		multigrid(mu,mf,md,mv,N_MALLAS,def_fila_g,def_fila,&nd);
		printf("Iteracion %d \t max(defecto)=%f\tratio=%0.10f\n",i, nd,nd/nda);
	}


	/* Liberacion de memoria */
	printf("Liberando memoria...\n");
	for(i=0;i<N_MALLAS;i++)
	{
		hipFree(mu[i].v);
		hipFree(mv[i].v);
		hipFree(mf[i].v);
		hipFree(md[i].v);
	}
	hipFree(def_fila_g);
	free(def_fila);
	free(mu);
	free(mv);
	free(mf);
	free(md);

}
void multigrid(Malla * u, Malla * f, Malla * v, Malla * d, int n_mallas, double * def_fila_g, double * def_fila, double *nd)
{
	int m = n_mallas -1; // m representa en la malla en la que estamos
	int dim;
	int dim_;
	int i;
	if(m==0) // estamos en la malla 0, buscamos la solución exacta
	{
		soluciona<<<1,1>>>(u[m],f[m]);
	}
	else
	{
		dim = (int)pow(2.0,m+1)+1;
		dim_= (int)pow(2.0,m)+1;

		/* Datos para llamar a la GPU */
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid(((int)dim+BLOCK_SIZE-1)/dimBlock.x,((int)dim+BLOCK_SIZE-1)/dimBlock.y);
		dim3 dimGrid_(((int)dim_+BLOCK_SIZE-1)/dimBlock.x,((int)dim_+BLOCK_SIZE-1)/dimBlock.y);


		/* Ponemos las mallas necesarias a 0 */
		cero<<<dimGrid,dimBlock>>>(d[m]);
		cero<<<dimGrid,dimBlock>>>(v[m]);
		cero<<<dimGrid_,dimBlock>>>(u[m-1]);
		cero<<<dimGrid_,dimBlock>>>(f[m-1]);


		/* Presuavizado */
		suavizado_n<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado rojo
		suavizado_n<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado rojo
		suavizado_n<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado rojo
		suavizado_n<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado rojo





		/* Defecto */
		defecto<<<dimGrid,dimBlock>>>(u[m],f[m],d[m]);

		/* Restringimos el defecto (de d a la f del nivel inferior)*/
		restringe<<<dimGrid_,dimBlock>>>(d[m],f[m-1]);

		/* Rellamada a multigrid */
		multigrid(u,f,d,v,m,def_fila_g,def_fila,nd);

		/* Imterpolamos (de la u de la malla inf a la v de la actual) */
		interpola<<<dimGrid_,dimBlock>>>(u[m-1],v[m]);

		/* Sumamos */
		suma<<<dimGrid,dimBlock>>>(u[m],v[m]);

		/* Volvemos a suavizar */
		suavizado_n<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado rojo
		suavizado_n<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado rojo
		suavizado_n<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado rojo
		suavizado_n<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u[m],f[m]); //Suavizado rojo





		/* En la malla mas fina, calculamos el defecto y comprobamos cómo va la cosa */
		if(m==N_MALLAS-1)
		{
			defecto<<<dimGrid,dimBlock>>>(u[m],f[m],d[m]);
			calcula_max<<<((int)dim+BLOCK_SIZE-1)/dimBlock.x,BLOCK_SIZE>>>(d[m],def_fila_g);
			hipMemcpy(def_fila,def_fila_g,dim*sizeof(double),hipMemcpyDeviceToHost);
			for(i=0;i<dim;i++)
			{
				if(abs(def_fila[i])>nd[0])
					nd[0]=abs(def_fila[i]);
			}
		}
	}
}

void imprime_g(Malla m) /* Imprime una malla allocada en la GPU */
{
	Malla m_h;	/* Definimos una malla, en el host */
	int i,j;
	m_h.v= (double *)malloc(m.dim*m.dim*sizeof(double));
	hipMemcpy(m_h.v,m.v,m.dim*m.dim*sizeof(double),hipMemcpyDeviceToHost);
	for(i=0;i<m.dim;i++)
	{
		for(j=0;j<m.dim;j++)
		{
			printf("%.4f ",m_h.v[i*m.dim+j]);
		}
		printf("\n");
	}
	free(m_h.v);
	printf("\n");
}

