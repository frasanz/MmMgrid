
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 16
#define DIM 4096 
__global__ void calcula_max(double * max)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	if(x<DIM)
	{
			max[x]=x;
	}
}

int main()
{
	int i;
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	double * g;
	double * h;
	hipMalloc((void**)&g,DIM*sizeof(double));
	h=(double *)malloc(DIM*sizeof(double));
	calcula_max<<<(DIM+BLOCK_SIZE-1)/dimBlock.x,BLOCK_SIZE>>>(g);
	hipMemcpy(h,g,DIM*sizeof(double),hipMemcpyDeviceToHost);
	for(i=0;i<DIM;i++)
		printf("%d %f\n",i,h[i]); 

	hipFree(g);
	free(h);
}
