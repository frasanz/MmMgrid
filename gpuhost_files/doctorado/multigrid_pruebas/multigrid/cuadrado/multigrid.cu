#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <multigrid_kernel.cu>

#define BLOCK_SIZE 16
#define MALLAS 2
void multigrid(Malla, Malla, int,double *);
void imprime_g(Malla);
void imprime_g2(Malla);

double anterior;

int main(int argc, char **argv)
{
	int m = MALLAS-1; /* m representa la malla en la que estamos */
	double dim  = pow(2.0,m+1)+1;
	double dim_ = pow(2.0,m)+1;
	double nd; //Norma del defecto
	double nda;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(((int)dim+BLOCK_SIZE-1)/dimBlock.x,((int)dim+BLOCK_SIZE-1)/dimBlock.y);
	dim3 dimGrid_(((int)dim_+BLOCK_SIZE-1)/dimBlock.x,((int)dim_+BLOCK_SIZE-1)/dimBlock.y);
	printf("%d %d\n",dimGrid.x,dimGrid.y);
	//Definimos la malla de la función en la GPU
	Malla f_g;
	f_g.dim = (int)dim;
	hipMalloc((void**)&f_g.v,dim*dim*sizeof(double));
	inicializa_f<<<dimGrid, dimBlock>>>(f_g);
	//imprime_g(f_g);


	//Definimos la malla de la solución en la GPU
	Malla u_g;
	u_g.dim = (int)dim;
	hipMalloc((void**)&u_g.v,dim*dim*sizeof(double));
	cero<<<dimGrid,dimBlock>>>(u_g);
	//imprime_g(u_g);

	//Llamamos a multigrid
	for(int i=0;i<1;i++)
	{
		nda=nd;
		nd=0.0;
		printf("[Iteración #%d]\n",i);
		multigrid(u_g,f_g,MALLAS,&nd);
		printf("\t max(defecto)=%f\tratio=%0.10f\n",nd,nd/nda);
	}
	//Limpieza de la GPU
	hipFree(u_g.v);
	hipFree(f_g.v);

}

void imprime_g(Malla m) /* Imprime una malla allocada en la GPU */
{
	Malla m_h;	/* Definimos una malla, en el host */
	int i,j;
	m_h.v= (double *)malloc(m.dim*m.dim*sizeof(double));
	hipMemcpy(m_h.v,m.v,m.dim*m.dim*sizeof(double),hipMemcpyDeviceToHost);
	for(i=0;i<m.dim;i++)
	{
		for(j=0;j<m.dim;j++)
		{
			printf("%.4f ",m_h.v[i*m.dim+j]);
		}
		printf("\n");
	}
	free(m_h.v);
	printf("\n");
}
void imprime_g2(Malla m) /* Imprime un elemento de una malla allocada en GPU */
{
	Malla m_h;	/* Definimos una malla, en el host */
	m_h.v= (double *)malloc(m.dim*m.dim*sizeof(double));
	hipMemcpy(m_h.v,m.v,m.dim*m.dim*sizeof(double),hipMemcpyDeviceToHost);
	printf("%.10f ",m_h.v[10*m.dim+10]);
	free(m_h.v);
	printf("\n");
}


void multigrid(Malla u, Malla f, int mallas,double * nd)
{
	double * norma_defecto_g;
	double * norma_defecto;
	int i;

	int m = mallas-1; /* m representa la malla en la que estamos */
	double dim  = pow(2.0,m+1)+1;
	double dim_ = pow(2.0,m)+1;

	if(m==0) /* Nº total de mallas=1, luego estamos en la malla 0 */
	{
		soluciona<<<1,1>>>(u,f);
	}
	else
	{

			/* Datos para llamar a la GPU */
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid(((int)dim+BLOCK_SIZE-1)/dimBlock.x,((int)dim+BLOCK_SIZE-1)/dimBlock.y);
		dim3 dimGrid_(((int)dim_+BLOCK_SIZE-1)/dimBlock.x,((int)dim_+BLOCK_SIZE-1)/dimBlock.y);

		/* Cramos las mallas necesarias */
		Malla d_g; // Para almacenar el defecto 
		d_g.dim = (int)dim;
		hipMalloc((void**)&d_g.v,dim*dim*sizeof(double));
		cero<<<dimGrid,dimBlock>>>(d_g);
		Malla d__g; // Para almacenar el defecto restringido
		d__g.dim=(int)dim_;
		hipMalloc((void **)&d__g.v,dim_*dim_*sizeof(double));
		cero<<<dimGrid_,dimBlock>>>(d__g);
		Malla v_g;	// Para almacenar las v
		v_g.dim = (int) dim;
		hipMalloc((void**)&v_g.v,dim*dim*sizeof(double));
		cero<<<dimGrid,dimBlock>>>(v_g);
		Malla v__g; // Para almacenar la v restringida
		v__g.dim = (int) dim_;
		hipMalloc((void**)&v__g.v,dim_*dim_*sizeof(double));
		cero<<<dimGrid_,dimBlock>>>(v__g);

		/* Presuavizado */
		suavizado_n<<<dimGrid, dimBlock>>>(u,f); // Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u,f); // Suavizado rojo
		imprime_g(u);
		//printf("Malla %d\n",m); imprime_g2(u);


		/* Defecto */
		defecto<<<dimGrid,dimBlock>>>(u,f,d_g);
		imprime_g(d_g);
		
		/* Restringimos el defecto */
		restringe<<<dimGrid_,dimBlock>>>(d_g,d__g);
		imprime_g(d__g);

		/* Rellamada a multigrid */
		multigrid(v__g,d__g,mallas-1,NULL);
		imprime_g(v__g);

		/* Interpolamos la v */
		interpola<<<dimGrid_,dimBlock>>>(v__g,v_g);
		imprime_g(v_g);

		/* Sumamos */
		suma<<<dimGrid,dimBlock>>>(u,v_g);
		imprime_g(u);

		/* Volvemos a suavizar */
		suavizado_n<<<dimGrid, dimBlock>>>(u,f);	//Suavizado negro
		suavizado_r<<<dimGrid, dimBlock>>>(u,f);	//Suavizado rojo
		imprime_g(u);
		printf("Mallas %d\n",m); imprime_g2(u);


		/* En la malla mas fina, calculamos el defecto y comprobamos cómo va la cosa */
		if(mallas==MALLAS)
		{
			defecto<<<dimGrid,dimBlock>>>(u,f,d_g);
			imprime_g(d_g);

			hipMalloc((void**)&norma_defecto_g,u.dim*sizeof(double)); //Los primeros para la norma de cada fila 
			norma_defecto=(double *)malloc(u.dim*sizeof(double));

			printf("%d %d",((int)dim+BLOCK_SIZE-1)/dimBlock.x,BLOCK_SIZE);
			calcula_max2<<<((int)dim+BLOCK_SIZE-1)/dimBlock.x,BLOCK_SIZE>>>(norma_defecto_g);

			hipMemcpy(norma_defecto,norma_defecto_g,u.dim*sizeof(double),hipMemcpyDeviceToHost);
			for(i=0;i<u.dim;i++)
			{
				printf("%d %f\n",i,norma_defecto[i]);
				if(abs(norma_defecto[i])>nd[0])
					nd[0]=abs(norma_defecto[i]);
			}
			printf("%f\n",nd[0]);


			free(norma_defecto);
			hipFree(norma_defecto_g);
		}


		/* Liberamos recursos */
		hipFree(d_g.v);
		hipFree(d__g.v);
		hipFree(v_g.v);
		hipFree(v__g.v);
	}
}
