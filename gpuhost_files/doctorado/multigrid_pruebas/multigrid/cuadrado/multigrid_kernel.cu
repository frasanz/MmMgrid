#include "hip/hip_runtime.h"
#include <multigrid.h>
__global__ void cero(Malla m)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if(x<m.dim && y < m.dim)
	{
		m.v[x*m.dim+y]=0.0;
	}
}

__global__ void inicializa_f(Malla m)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	double coord_x=1.0*y/(m.dim-1);
	double coord_y=1.0*x/(m.dim-1);
	if(x==m.dim-1 || y==m.dim-1 || x==0 || y==0)
		m.v[x*m.dim+y]=0.0;
	if(x<m.dim-1 && y<m.dim-1 && x>0 && y>0)
		m.v[x*m.dim+y]=cos(coord_x*coord_y);
}


__global__ void inicializa(Malla m)
{

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if(x==m.dim-1 || y==m.dim-1 || x==0 || y==0)
		m.v[x*m.dim+y]=0.0;
	if(x<m.dim-1 && y<m.dim-1 && x>0 && y>0)
		m.v[x*m.dim+y]=1;
}
__global__ void suavizado_n(Malla u, Malla f)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	double h2=pow(1.0/(u.dim-1),2);
	if(x<u.dim-1 && y < u.dim-1 && x>0 && y>0 && !((x+y)%2))
		u.v[x*u.dim+y]=0.25*(f.v[x*u.dim+y]*h2+
		                     u.v[(x-1)*u.dim+y]+
												 u.v[(x+1)*u.dim+y]+
												 u.v[x*u.dim+y-1]+
												 u.v[x*u.dim+y+1]);
}
__global__ void suavizado_r(Malla u, Malla f)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	double h2=pow(1.0/(u.dim-1),2);
	if(x<u.dim-1 && y < u.dim-1 && x>0 && y>0 && ((x+y)%2))
		u.v[x*u.dim+y]=0.25*(f.v[x*u.dim+y]*h2+
		                     u.v[(x-1)*u.dim+y]+
												 u.v[(x+1)*u.dim+y]+
												 u.v[x*u.dim+y-1]+
												 u.v[x*u.dim+y+1]);
}
__global__ void defecto(Malla u, Malla f, Malla d)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	double h2=pow(1.0/(u.dim-1),2);
	if(x==u.dim-1 || y==u.dim-1 || x==0 || y==0)
		d.v[x*u.dim+y]=0.0;
	if(x<u.dim-1 && y < u.dim-1 && x>0 && y>0)
	{
		d.v[x*u.dim+y]=f.v[x*u.dim+y]-
			(4*u.v[x*u.dim+y]-u.v[(x-1)*u.dim+y]-u.v[(x+1)*u.dim+y]-u.v[x*u.dim+(y-1)]-u.v[x*u.dim+(y+1)])/h2;
	}
}


__global__ void restringe(Malla u, Malla u_)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if(x<u_.dim-1 && y<u_.dim-1 && x>0 && y>0)
	{
		u_.v[x*u_.dim+y]=(
			4*u.v[(2*x  )*u.dim+2*y  ]+
			2*u.v[(2*x-1)*u.dim+2*y  ]+
			2*u.v[(2*x+1)*u.dim+2*y  ]+
			2*u.v[(2*x  )*u.dim+2*y-1]+
			2*u.v[(2*x  )*u.dim+2*y+1]+
			  u.v[(2*x-1)*u.dim+2*y-1]+
				u.v[(2*x-1)*u.dim+2*y+1]+
				u.v[(2*x+1)*u.dim+2*y-1]+
				u.v[(2*x+1)*u.dim+2*y+1])/16;
	}

}

__global__ void interpola(Malla u_, Malla u)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if(x<u_.dim && y< u_.dim )
	{
		u.v[2*x*u.dim+2*y]=u_.v[x*u_.dim+y];
		if(2*x+1<u.dim)
			u.v[(2*x+1)*u.dim+2*y]=(u_.v[x*u_.dim+y]+u_.v[(x+1)*u_.dim+y])/2;
		if(2*y+1<u.dim)
			u.v[2*x*u.dim+2*y+1]=(u_.v[x*u_.dim+y]+u_.v[x*u_.dim+y+1])/2;
		if(2*x+1<u.dim && 2*y+1<u.dim)
			u.v[(2*x+1)*u.dim+2*y+1]=(u_.v[x*u_.dim+y]  +u_.v[(x+1)*u_.dim+y]+
			                          u_.v[x*u_.dim+y+1]+u_.v[(x+1)*u_.dim+y+1])/4;


	}
}
__global__ void suma(Malla u, Malla v)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if(x<u.dim && y< u.dim)
	{
		u.v[x*u.dim+y]=u.v[x*u.dim+y]+v.v[x*u.dim+y];
	}
}
__global__ void soluciona(Malla u, Malla f)
{
	u.v[4]=f.v[4]/16;
}

__global__ void calcula_max(Malla m, double * max)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int j;
	max[x]=0.0;
	if(x<m.dim)
	{
		for(j=1;j<m.dim-1;j++)
		{
			max[x]=x;
			if(abs(m.v[x*m.dim+j])>max[x])
				max[x]=x;
		}
	}
}
__global__ void calcula_max2(double * max)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	if(x<257)
	{
			max[x]=x;
	}
}


