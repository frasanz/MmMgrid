/*
 * =====================================================================================
 *
 *       Filename:  jacobi_cpu.c
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/11 02:30:51
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *        Company:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define SIZE 8192
#define BLOCK_SIZE 16

float ratio(float**u,float ant,int iter)
{
	float tmp=0.0;
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			if(u[i][j]>tmp)
				tmp=u[i][j];
		}
	}
	if(iter%10==0)
		printf(" iter=%d ratio=%f max=%f\n",iter,tmp/ant,tmp);
	return tmp;
}

void muestra(float**u)
{
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%f ",u[i][j]);
		}
		printf("\n");
	}
}
__global__ void jacobi(float *d_u_new,float *d_u, float *d_f,float h, float pr)
{

	__shared__ float bf[BLOCK_SIZE][BLOCK_SIZE];

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	int k = threadIdx.x;
	int l = threadIdx.y;
	
	bf[k][l] = d_u[i*SIZE+j];

	__syncthreads();

	if(i>=0 && i < SIZE && j>=0 && j<SIZE)
	{
		d_u_new[i*SIZE+j]=0.25*(
											h*h*d_f[i    *SIZE+j     ]+
											      bf[k-1][l]+
														bf[k+1][l]+
														bf[k][l-1]+
														bf[k][l+1]);
		d_u_new[i*SIZE+j]=pr;
	}
}
int main()
{
	float * h_u, * h_f;
	float * d_u, *d_u_new, *d_f;
	float * temp;
	float ant=1.0;
	int i,j;
	size_t size;
	float h = 1.0/SIZE;


	/* Reservamos memoria */
	size=SIZE*SIZE*sizeof(float);
	printf("Necesitamos %d Mb\n",3*size/1024/1024);
	h_u     = (float*)malloc(size);
	h_f     = (float*)malloc(size);

	/* REservamos memoria GPU*/
	hipMalloc(&d_u,size);
	hipMalloc(&d_u_new,size);
	hipMalloc(&d_f,size);


	/* Inicializamos */
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			h_f[i*SIZE+j]=0.0;
			h_u[i*SIZE+j]=rand();
		}
	}
	for(i=0;i<SIZE;i++)
	{
		h_u[i]=0.0;
		h_u[i*SIZE]=0.0;
		h_u[SIZE*(SIZE-1)+i]=0.0;
		h_u[i*SIZE+SIZE-1]=0.0;
	}

	/* Copiamos la memoria del host a la GPU */
	hipMemcpy(d_f,h_f,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u,h_u,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u_new,h_u,size,hipMemcpyHostToDevice);

	/* Creamos el grid para el cálculo */
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);

	/* Bucle principal, llamamos a JACOBI */
	for(i=0;i<100;i++)
	{
		jacobi<<<dimGrid,dimBlock>>>(d_u_new,d_u,d_f,h,1.0*i);
		if(i%10==0) 
			printf("iter=%d\n",i); 
	}

	/* Copiamos */
	hipMemcpy(h_u,d_u_new,size,hipMemcpyDeviceToHost);

	/* Comprobamos */
	for(i=0;i<SIZE*SIZE;i++)
		if(h_u[i]!=1.0*99)
			printf("Error en %d, h_u[%d]=%f\n",i,i,h_u[i]);

	/* Liberamos memoria */
	free(h_u);
	free(h_f);
	hipFree(d_u_new);
	hipFree(d_u);
	hipFree(d_f);
}

