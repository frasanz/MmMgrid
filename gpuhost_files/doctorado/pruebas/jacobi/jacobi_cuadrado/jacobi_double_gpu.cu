/*
 * =====================================================================================
 *
 *       Filename:  jacobi_cpu.c
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/11 02:30:51
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *        Company:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define SIZE 8192
#define BLOCK_SIZE 16

double ratio(double**u,double ant,int iter)
{
	double tmp=0.0;
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			if(u[i][j]>tmp)
				tmp=u[i][j];
		}
	}
	if(iter%10==0)
		printf(" iter=%d ratio=%f max=%f\n",iter,tmp/ant,tmp);
	return tmp;
}

void muestra(double**u)
{
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%f ",u[i][j]);
		}
		printf("\n");
	}
}
__global__ void jacobi(double *d_u_new,double *d_u, double *d_f,double h, double pr)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if(i>=0 && i < SIZE && j>=0 && j<SIZE)
	{
		d_u_new[i*SIZE+j]=0.25*(
											h*h*d_f[i    *SIZE+j     ]+
											      d_u[(i-1)*SIZE+j     ]+
														d_u[(i+1)*SIZE+j     ]+
														d_u[i    *SIZE+j-1   ]+
														d_u[i    *SIZE+j+1   ]);
		d_u_new[i*SIZE+j]=pr;
	}
}
int main()
{
	double * h_u, * h_f;
	double * d_u, *d_u_new, *d_f;
	double * temp;
	double ant=1.0;
	int i,j;
	size_t size;
	double h = 1.0/SIZE;


	/* Reservamos memoria */
	size=SIZE*SIZE*sizeof(double);
	printf("Necesitamos %d Mb\n",3*size/1024/1024);
	h_u     = (double*)malloc(size);
	h_f     = (double*)malloc(size);

	/* REservamos memoria GPU*/
	hipMalloc(&d_u,size);
	hipMalloc(&d_u_new,size);
	hipMalloc(&d_f,size);


	/* Inicializamos */
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			h_f[i*SIZE+j]=0.0;
			h_u[i*SIZE+j]=rand();
		}
	}
	for(i=0;i<SIZE;i++)
	{
		h_u[i]=0.0;
		h_u[i*SIZE]=0.0;
		h_u[SIZE*(SIZE-1)+i]=0.0;
		h_u[i*SIZE+SIZE-1]=0.0;
	}

	/* Copiamos la memoria del host a la GPU */
	hipMemcpy(d_f,h_f,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u,h_u,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u_new,h_u,size,hipMemcpyHostToDevice);

	/* Creamos el grid para el cálculo */
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);

	/* Bucle principal, llamamos a JACOBI */
	for(i=0;i<700;i++)
	{
		jacobi<<<dimGrid,dimBlock>>>(d_u_new,d_u,d_f,h,1.0*i);
		if(i%10==0) 
			printf("iter=%d\n",i); 
	}

	/* Copiamos */
	hipMemcpy(h_u,d_u_new,size,hipMemcpyDeviceToHost);

	/* Comprobamos */
	for(i=0;i<SIZE*SIZE;i++)
		if(h_u[i]!=1.0*699)
			printf("Error en %d, h_u[%d]=%f\n",i,i,h_u[i]);

	/* Liberamos memoria */
	free(h_u);
	free(h_f);
	hipFree(d_u_new);
	hipFree(d_u);
	hipFree(d_f);
}

