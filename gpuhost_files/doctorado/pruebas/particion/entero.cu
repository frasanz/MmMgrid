/*
 * =====================================================================================
 *
 *       Filename:  jacobi_cpu.c
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/11 02:30:51
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *        Company:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define SIZE 8192
#define BLOCK_SIZE 32

float ratio(float**u,float ant,int iter)
{
	float tmp=0.0;
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			if(u[i][j]>tmp)
				tmp=u[i][j];
		}
	}
	if(iter%10==0)
		printf(" iter=%d ratio=%f max=%f\n",iter,tmp/ant,tmp);
	return tmp;
}

void muestra(float**u)
{
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%f ",u[i][j]);
		}
		printf("\n");
	}
}

__global__ void jacobi(float *d_u_new,float *d_u, float *d_f,float h,float val)
{
	__shared__ float bf[BLOCK_SIZE][BLOCK_SIZE];

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = threadIdx.x;
	int l = threadIdx.y;

	bf[k][l] = d_u[i*SIZE+j];

	__syncthreads();

	if(i>0 && j>0 && i<SIZE-1 && j<SIZE-1)
	d_u_new[i*SIZE+j]=0.25*bf[k][l];
	d_u_new[i*SIZE+j]=1.0;
}
int main()
{
	float * h_u, * h_f;
	float * d_u, *d_u_new, *d_f;
	float * temp;
	float suma=0.0;
	int i,j;
	size_t size;
	float h = 1.0/SIZE;


	/* Reservamos memoria */
	size=SIZE*SIZE*sizeof(float);
	printf("Necesitamos %d Mb\n",3*size/1024/1024);
	h_u     = (float*)malloc(size);
	h_f     = (float*)malloc(size);

	/* REservamos memoria GPU*/
	hipMalloc(&d_u,size);
	hipMalloc(&d_u_new,size);
	hipMalloc(&d_f,size);


	/* Inicializamos */
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			h_f[i*SIZE+j]=0.0;
			h_u[i*SIZE+j]=1.0*i;
		}
	}
	for(i=0;i<SIZE;i++)
	{
		h_u[i]=0.0;
		h_u[i*SIZE]=0.0;
		h_u[SIZE*(SIZE-1)+i]=0.0;
		h_u[i*SIZE+SIZE-1]=0.0;
	}

	/* Copiamos la memoria del host a la GPU */
	hipMemcpy(d_f,h_f,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u,h_u,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u_new,h_u,size,hipMemcpyHostToDevice);

	/* Creamos el grid para el cálculo */
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);

	/* Bucle principal, llamamos a JACOBI */
	for(i=0;i<1000;i++)
	{
		jacobi<<<dimGrid,dimBlock>>>(d_u_new,d_u,d_f,h,1.0*i);
		temp=d_u;
		d_u=d_u_new;
		d_u_new=temp;
		if(i%100==0)
		{
			printf("iter=%d\n",i);
			printf("Copiando de GPU a CPU\n");
			hipMemcpy(h_u,d_u,size,hipMemcpyDeviceToHost);
			printf(" %f %f %f\n",  h_u[100], h_u[101],h_u[102]);



		}
	}

	/* Copiamos la memoria de la GPU al host */
	printf("Copiando de GPU a CPU\n");

	hipMemcpy(h_u,d_u,size,hipMemcpyDeviceToHost);


	printf("Sumando en cpu\n");
	/* Sumamos todos los elementos, una especie de checksum */
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			suma+=h_u[i*SIZE+j];
		}
	}

	printf("%f %f %f %f\n", suma, h_u[100], h_u[101],h_u[102]);


	/* Liberamos memoria */
	free(h_u);
	free(h_f);
	hipFree(d_u_new);
	hipFree(d_u);
	hipFree(d_f);
}

