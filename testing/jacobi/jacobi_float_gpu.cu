/*
 * =====================================================================================
 *
 *       Filename:  jacobi_cpu.c
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/11 02:30:51
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *        Company:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define SIZE 2048
#define BLOCK_SIZE 16

float ratio(float**u,float ant,int iter)
{
	float tmp=0.0;
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			if(u[i][j]>tmp)
				tmp=u[i][j];
		}
	}
	if(iter%10==0)
		printf(" iter=%d ratio=%f max=%f\n",iter,tmp/ant,tmp);
	return tmp;
}

void muestra(float**u)
{
	int i,j;
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%f ",u[i][j]);
		}
		printf("\n");
	}
}
__global__ void jacobi(float *d_u_new,float *d_u, float *d_f,float h)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	d_u_new[i*SIZE+j]=0.25*(
			h*h*d_f[i    *SIZE+j     ]+
			d_u[(i-1)*SIZE+j     ]+
			d_u[(i+1)*SIZE+j     ]+
			d_u[i    *SIZE+j-1   ]+
			d_u[i    *SIZE+j+1   ]);
}
int main()
{
	float * h_u, * h_f;
	float * d_u, *d_u_new, *d_f;
	float * temp;
	float ant=1.0;
	int i,j;
	size_t size;
	float h = 1.0/SIZE;


	/* Reservamos memoria */
	size=SIZE*SIZE*sizeof(float);
	printf("Necesitamos %d Mb\n",3*size/1024/1024);
	h_u     = (float*)malloc(size);
	h_f     = (float*)malloc(size);

	/* REservamos memoria GPU*/
	hipMalloc(&d_u,size);
	hipMalloc(&d_u_new,size);
	hipMalloc(&d_f,size);


	/* Inicializamos */
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			h_f[i*SIZE+j]=0.0;
			h_u[i*SIZE+j]=rand();
		}
	}
	for(i=0;i<SIZE;i++)
	{
		h_u[i]=0.0;
		h_u[i*SIZE]=0.0;
		h_u[SIZE*(SIZE-1)+i]=0.0;
		h_u[i*SIZE+SIZE-1]=0.0;
	}

	/* Copiamos la memoria del host a la GPU */
	hipMemcpy(d_f,h_f,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u,h_u,size,hipMemcpyHostToDevice);
	hipMemcpy(d_u_new,h_u,size,hipMemcpyHostToDevice);

	/* Creamos el grid para el cálculo */
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(SIZE/BLOCK_SIZE,SIZE/BLOCK_SIZE);

	/* Bucle principal, llamamos a JACOBI */
	for(i=0;i<300;i++)
	{
		jacobi<<<dimGrid,dimBlock>>>(d_u_new,d_u,d_f,h);
		if(i%10==0)
			printf("iter=%d\n",i);
	}

	/* Liberamos memoria */
	free(h_u);
	free(h_f);
	hipFree(d_u_new);
	hipFree(d_u);
	hipFree(d_f);
}

